#include "hip/hip_runtime.h"

/*******************************************************************************
 * (c) 2019 Paul Gordon's parallel (CUDA) NVIDIA GPU implementation of the Dynamic Time 
 * Warp Barycenter Averaging algorithm as conceived (without parallel compuation conception) by Francois Petitjean 
 ******************************************************************************/

#include <string.h>
#include <iostream>
#include <fstream>
#include "cpu_utils.hpp"
#include "dba.hpp"

template<typename T>
void
setupAndRun(char **series_file_names, int num_series, char *output_prefix, int binary_read_mode, int use_open_start, int use_open_end, double convergence_delta){
	size_t *sequence_lengths = 0;
	size_t averageSequenceLength = 0;
	void *averageSequence = 0;
	T **sequences = 0;
        if(binary_read_mode){ readSequenceBinaryFiles<T>(series_file_names, num_series, &sequences, &sequence_lengths); }
        else{ readSequenceTextFiles<T>(series_file_names, num_series, &sequences, &sequence_lengths); }

	// Shorten sequence names to everything before the first "." in the file name
	for (int i = 0; i < num_series; i++){ char *z = strchr(series_file_names[i], '.'); if(z) *z = '\0';}
        performDBA<T>(sequences, num_series, sequence_lengths, series_file_names, convergence_delta, use_open_start, use_open_end, output_prefix, (T **) &averageSequence, &averageSequenceLength);

	std::ofstream avg_file((std::string(output_prefix)+std::string(".avg.txt")).c_str());
        for (size_t i = 0; i < averageSequenceLength; ++i) { avg_file << ((T *) averageSequence)[i] << std::endl; }
	avg_file.close();

	// Cleanup
        for (int i = 0; i < num_series; i++){ hipHostFree(sequences[i]); }
        hipHostFree(sequences);
	hipHostFree(sequence_lengths); CUERR("Freeing CPU memory for the sequence lengths");
	hipHostFree(averageSequence); CUERR("Freeing CPU memory for the DBA result");
}

__host__
int main(int argc, char **argv){

	if(argc < 8){
#if DOUBLE_UNSUPPORTED == 1
		std::cout << "Usage: " << argv[0] << " <binary|text> <int|uint|ulong|float> " <<
#else
		std::cout << "Usage: " << argv[0] << " <binary|text> <int|uint|ulong|float|double> " <<
#endif
		          "<global|open_start|open_end|open> <output files prefix> <delta criterium for convergence, in range (0,1]> <series1.txt> <series2.txt> [series3.txt...]\n";
		exit(1);
     	}

	int num_series = argc-6;
	double convergence_delta = atof(argv[4]);
	if(convergence_delta <= 0.0 || convergence_delta > 1){
		std::cerr << "Fourth argument (" << argv[3] << ") could not be parsed into a number in the acceptable range (0,1]" << std::endl;
		exit(1);
	} 
	int binary_read_mode = 0;
	if(!strcmp(argv[1],"binary")){
		binary_read_mode = 1;
	}
	else if(strcmp(argv[1],"text")){
		std::cerr << "First argument (" << argv[1] << ") is neither 'binary' nor 'text'" << std::endl;
		exit(1);
	}

	int use_open_start = 0;
	int use_open_end = 0;
	if(!strcmp(argv[3],"global")){
        }
        else if(!strcmp(argv[3],"open_start")){
		use_open_start = 1;
        }
        else if(!strcmp(argv[3],"open_end")){
		use_open_end = 1;
        }
	else if(strcmp(argv[3],"open")){
		use_open_start = 1;
		use_open_end = 1;
        }
	else{
		std::cerr << "Third argument (" << argv[3] << ") is not one of the accept values 'global', 'open_start', 'open_end' or 'open'" << std::endl;
                exit(1);
	}

	char *output_prefix = argv[5];

	int argind = 6; // Where the file names start
	// The following are all the data types supported by CUDA's atomicAdd() operation, so we support them too for best value precision maintenance.
	if(!strcmp(argv[2],"int")){
		setupAndRun<int>(&argv[argind], num_series, output_prefix, binary_read_mode, use_open_start, use_open_end, convergence_delta);
	}
	else if(!strcmp(argv[2],"uint")){
		setupAndRun<unsigned int>(&argv[argind], num_series, output_prefix, binary_read_mode, use_open_start, use_open_end, convergence_delta);
	}
	else if(!strcmp(argv[2],"ulong")){
		setupAndRun<unsigned long long>(&argv[argind], num_series, output_prefix, binary_read_mode, use_open_start, use_open_end, convergence_delta);
	}
	else if(!strcmp(argv[2],"float")){
		setupAndRun<float>(&argv[argind], num_series, output_prefix, binary_read_mode, use_open_start, use_open_end, convergence_delta);
	}
	// Only since CUDA 6.1 (Pascal and later architectures) is atomicAdd(double *...) supported.  Remove if you want to compile for earlier graphics cards.
#if DOUBLE_UNSUPPORTED == 1
#else
	else if(!strcmp(argv[2],"double")){
		setupAndRun<double>(&argv[argind], num_series, output_prefix, binary_read_mode, use_open_start, use_open_end, convergence_delta);
	}
#endif
	else{
		std::cerr << "Second argument (" << argv[2] << ") was not one of the accepted numerical representations: 'int', 'uint', 'ulong', 'float' or 'double'" << std::endl;
		exit(1);
	}

	// Following needed to allow cuda-memcheck to detect memory leaks
	hipDeviceReset(); CUERR("Resetting GPU device");
}
